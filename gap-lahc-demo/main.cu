#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>


#include "gpulib/types.h"
#include "gpulib/gpu.cuh"

#include "Instance.h"
#include "Solution.h"
#include "gSolution.cuh"
#include "guloso.h"

const int nThreads = 10;

int main(int argc, char *argv[]){
//int main(){
	struct timeval inicio;
	struct timeval fim;
	int tmili;
	const char *fileName = argv[1];
	//const char *fileName = "d05100";
	int deviceCount = 0;
	//int i;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(1);
	}
	if(deviceCount == 0)
	{
		printf("No GPU found :(");
		exit(1);
	}
	else
	{
		printf("Found %d GPUs!\n", deviceCount);
		gpuSetDevice(0);
		printf("GPU 0 initialized!\n");
	}

	Instance *d_instance;
	Solution *d_solution;
	hiprandState_t* states;
	hipMalloc((void**) &states, nThreads * sizeof(hiprandState_t));


	Instance *inst = loadInstance(fileName);
	Solution *sol = allocationPointersSolution(inst);
	if(fileName[0]=='e'){
		sol = guloso(inst,1,20);
	}else{
		sol = guloso(inst,1,2);
	}

	showInstance(inst);
	printf("Load data instance ok!\n");
	getchar();
	showSolution(sol,inst);
	printf("greedy solution ok!\n");
	getchar();
	srand(time(NULL));
	//for(int i=0;i<=10;i++){
	//schc_cpu(sol, inst, 50);
	//}
	//getchar();

	d_instance = createGPUInstance(inst, inst->nJobs, inst->mAgents);
	d_solution = createGPUsolution(sol,inst->nJobs, inst->mAgents);


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	gettimeofday(&inicio, NULL);
	//schc_cpu(sol,inst,100);
	hipEventRecord(start);

	SCHC<<<1,nThreads>>>(d_instance,d_solution, time(NULL), states, 100);
	gpuDeviceSynchronize();
	hipEventRecord(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("time: %.4fms\n", milliseconds);
	gettimeofday(&fim, NULL);
	tmili = (int) (1000 * (fim.tv_sec - inicio.tv_sec) + (fim.tv_usec - inicio.tv_usec) / 1000);
	printf("tempo: %d\n",tmili);
	gpuFree(d_instance);
	gpuFree(d_solution);
	free(inst);
	free(sol);
	printf("program finished successfully!\n");
	return 0;
}
