#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>


#include "gpulib/types.h"
#include "gpulib/gpu.cuh"

#include "Instance.h"
#include "Solution.h"
#include "gSolution.cuh"
#include "guloso.h"

const int nThreads = 576;
const int nBlocks = 28;
int main(int argc, char *argv[]){
	//counter
	int i,j;
	
	
	//Position and Best Solution
	int pos_best;
	int cost_best;
	
	//Parameters of heuristic SCHC
	int l_c=0;
	
	//Variable with size of struct solution
	size_t size_solution;
	
	//File name of instance GAP
	const char *fileName = argv[1];
	
	//Variable with numbers of GPU's
	int deviceCount = 0;
	
	//Commands for verify use correct of GPU
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(1);
	}
	if(deviceCount == 0)
	{
		printf("No GPU found :(");
		exit(1);
	}
	else
	{
		printf("Found %d GPUs!\n", deviceCount);
		gpuSetDevice(0);
		printf("GPU 0 initialized!\n");
	}
	
	//Pointer of instance and solution for use in GPU (device)
	Instance *d_instance;
	Solution *d_solution;
	
	//Pointer of states for use with hiprand
	hiprandState_t* states;
	hipMalloc((void**) &states, (nThreads*nBlocks) * sizeof(hiprandState_t));
	
	//Pointer of seed for use with hiprand (host)
	unsigned int *h_seed = (unsigned int*)malloc(sizeof(unsigned int)*(nThreads*nBlocks));
	srand(time(NULL));
	for(i=0;i<(nThreads*nBlocks);i++){
		h_seed[i] = rand()%100000;
	}
	
	//Pointer of instance and solution for use in GPU (device)
	Instance *inst = loadInstance(fileName); // Load the Instance 
	Solution *sol;
	
	
	//Generate of solution initial with greedy heuristic
	if(fileName[0]=='e'){
		sol = guloso(inst,1,20);
	}else{
		sol = guloso(inst,1,2);
	}
	
	//Definy of Solution size
	size_solution = sizeof(Solution)
							+ sizeof(TcostFinal)*nBlocks
							+ sizeof(Ts)*(inst->nJobs*nBlocks) //vector s
							+ sizeof(TresUsage)*(inst->mAgents*nBlocks); //vector resUsage
	
	//Reallocation of pointers Instance and Solution for GPU (device)
	d_instance = createGPUInstance(inst, inst->nJobs, inst->mAgents);
	d_solution = createGPUsolution(sol,inst->nJobs, inst->mAgents);
	
	//Pointer of rank in host, use for compute frequency of solution
	unsigned int *h_rank = (unsigned int*)malloc(sizeof(unsigned int)*inst->nJobs*inst->mAgents);
	memset(h_rank,0,sizeof(unsigned int)*inst->nJobs*inst->mAgents);
	
	//Pointers seed and rank in device (GPU)
	unsigned int *d_rank;
	unsigned int *d_seed;
	
	//Event and gpu for contability time 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Allocation of pointer and copy value in d_seed (Device)
	gpuMalloc((void*)&d_seed, sizeof(unsigned int)*(nThreads*nBlocks));
	gpuMemcpy(d_seed, h_seed, sizeof(unsigned int)*(nThreads*nBlocks), hipMemcpyHostToDevice);

	// Allocation of pointer and copy value in d_rank (Device)
	gpuMalloc((void* ) &d_rank, sizeof(unsigned int)*inst->nJobs*inst->mAgents);
	gpuMemcpy(d_rank, h_rank,sizeof(unsigned int)*inst->nJobs*inst->mAgents , hipMemcpyHostToDevice);
	
	
	//int blockSize;      // The launch configurator returned block size 
	//int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
	//int gridSize;
	//int N = 1000000;
	
	//hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, SCHC, 0, N);
	
	//printf("block size %d\n",blockSize);
	//printf("Min Grid %d\n",minGridSize);
	
	//Initial count time
	hipEventRecord(start);
	
	//Execute kernell of SCHC
	SCHC<<<nBlocks,nThreads>>>(d_instance,d_solution, d_seed ,d_rank, states, l_c);

	//Final count time
	hipEventRecord(stop);

	
	//copy solution of device to host
	gpuMemcpy(sol, d_solution, size_solution, hipMemcpyDeviceToHost);
	
	//copy rank (frequency) of device to host
	gpuMemcpy(h_rank, d_rank,sizeof(unsigned int)*inst->nJobs*inst->mAgents , hipMemcpyDeviceToHost);
	
	//syncronize of output GPU
	hipEventSynchronize(stop);
	
	//Compute time of execution in kernel GPU
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%.4fms\n", milliseconds);

	
	
	//reallocation pointers of Instance
	inst->cost = (Tcost*)(inst+1);
	inst->resourcesAgent =(TresourcesAgent*) (inst->cost +(inst->nJobs*inst->mAgents));
	inst->capacity =(Tcapacity*) (inst->resourcesAgent + (inst->nJobs*inst->mAgents));

	//reallocation pointers of Solution
	sol->costFinal = (TcostFinal*)(sol+1);
	sol->s = (Ts*)(sol->costFinal + nBlocks);
	sol->resUsage = (TresUsage*)(sol->s + (inst->nJobs*nBlocks));
	pos_best=0;
	cost_best  = sol->costFinal[0];
	for(i=1;i<nBlocks;i++){
		printf("cost final: %d\n",sol->costFinal[i]);
		if(sol->costFinal[i]<cost_best){
			pos_best = i;
			cost_best = sol->costFinal[i]; 
		}
	}
	
	printf("Cost best solution: %d\n",cost_best);
	for(i=0;i<inst->nJobs;i++){
		printf("%d ", sol->s[i+inst->nJobs*pos_best]+1);
		
	}
	printf("\n");
	
	//Create file .dat for use in LP
	createDat(inst, h_rank, fileName);
	create_solution(sol,inst,pos_best,fileName);
	create_frequency(inst, h_rank, fileName);
	//Free memory allocated
	gpuFree(d_instance);
	gpuFree(d_solution);
	gpuFree(d_rank);
	gpuFree(d_seed);
	gpuFree(states);
	free(inst);
	free(sol);

	
	return 0;
}
