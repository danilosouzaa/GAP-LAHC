#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "gpulib/types.h"
#include "gpulib/gpu.cuh"

#include "Instance.h"

__global__ void teste(Instance *inst){
	printf("number of jobs: %d \n",inst->nJobs);

}

int main(){
	const char *fileName = "a05100";

	int deviceCount = 0;
	//int i;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(1);
	}
	if(deviceCount == 0)
	{
		printf("No GPU found :(");
		exit(1);
	}
	else
	{
		printf("Found %d GPUs!\n", deviceCount);
		gpuSetDevice(0);
		printf("GPU 0 initialized!\n");
	}

	Instance *d_instance;

	Instance *inst = loadInstance(fileName);
	showInstance(inst);
	printf("Load data instance ok!\n");
	getchar();

	d_instance = createGPUInstance(inst, inst->nJobs, inst->mAgents);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	teste<<<1,1>>>(d_instance);
	hipEventRecord(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("time: %.4fms\n", milliseconds);

	gpuFree(d_instance);
	free(inst);
	printf("program finished successfully!\n");
	return 0;
}
