#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "gpulib/types.h"
#include "gpulib/gpu.cuh"

#include "Instance.h"
#include "Solution.h"
#include "gSolution.cuh"
#include "guloso.h"


int main(int argc, char *argv[]){
//int main(){
	const char *fileName = argv[1];
	//const char *fileName = "b05100";
	int deviceCount = 0;
	//int i;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(1);
	}
	if(deviceCount == 0)
	{
		printf("No GPU found :(");
		exit(1);
	}
	else
	{
		printf("Found %d GPUs!\n", deviceCount);
		gpuSetDevice(0);
		printf("GPU 0 initialized!\n");
	}

	Instance *d_instance;
	Solution *d_solution;
	hiprandState_t* states;
	hipMalloc((void**) &states, 10 * sizeof(hiprandState_t));


	Instance *inst = loadInstance(fileName);
	printf("teste\n");
	Solution *sol = allocationPointersSolution(inst);
	sol = guloso(inst,1,2);
	showInstance(inst);
	printf("Load data instance ok!\n");
	getchar();
	showSolution(sol,inst);
	printf("greedy solution ok!\n");
	getchar();
	srand(time(NULL));
	//for(int i=0;i<=10;i++){
		//schc_cpu(sol, inst, 50);
	//}
	//getchar();

	d_instance = createGPUInstance(inst, inst->nJobs, inst->mAgents);
	d_solution = createGPUsolution(sol,inst->nJobs, inst->mAgents);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	hipEventRecord(start);
	SCHC<<<1,10>>>(d_instance,d_solution, time(NULL), states, 100);

	hipEventRecord(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("time: %.4fms\n", milliseconds);

	gpuFree(d_instance);
	gpuFree(d_solution);
	free(inst);
	free(sol);
	printf("program finished successfully!\n");
	return 0;
}
