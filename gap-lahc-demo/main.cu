#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "gpulib/types.h"
#include "gpulib/gpu.cuh"

#include "Instance.h"

__global__ void teste(Instance *inst, unsigned int seed, hiprandState_t* states){
	int aux;
	hiprand_init(seed,threadIdx.x,0,&states[threadIdx.x]);
	if(threadIdx.x < 1){
		//aux = hiprand(&states[blockIdx.x])%10;
		printf("number of jobs: %d \n",inst->nJobs);
		printf("Valor randomico : %d \n", hiprand(&states[threadIdx.x])%10);
		printf("Valor randomico : %d \n", hiprand(&states[threadIdx.x])%10);
	}

}

int main(){
	const char *fileName = "a05100";

	int deviceCount = 0;
	//int i;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(1);
	}
	if(deviceCount == 0)
	{
		printf("No GPU found :(");
		exit(1);
	}
	else
	{
		printf("Found %d GPUs!\n", deviceCount);
		gpuSetDevice(0);
		printf("GPU 0 initialized!\n");
	}

	Instance *d_instance;
	hiprandState_t* states;
	hipMalloc((void**) &states, 2 * sizeof(hiprandState_t));


	Instance *inst = loadInstance(fileName);
	showInstance(inst);
	printf("Load data instance ok!\n");
	getchar();

	d_instance = createGPUInstance(inst, inst->nJobs, inst->mAgents);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	teste<<<1,2>>>(d_instance, time(NULL), states);
	hipEventRecord(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("time: %.4fms\n", milliseconds);

	gpuFree(d_instance);
	free(inst);
	printf("program finished successfully!\n");
	return 0;
}
