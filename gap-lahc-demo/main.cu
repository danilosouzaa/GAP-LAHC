#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>


#include "gpulib/types.h"
#include "gpulib/gpu.cuh"

#include "Instance.h"
#include "Solution.h"
#include "gSolution.cuh"
#include "guloso.h"

const int nThreads = 1024;
int main(int argc, char *argv[]){
//int main(){
	struct timeval inicio;
	struct timeval fim;
	int tmili;
	size_t size_solution;
	const char *fileName = argv[1];
	//const char *fileName = "a05100";
	int deviceCount = 0;
	int i,j;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(1);
	}
	if(deviceCount == 0)
	{
		printf("No GPU found :(");
		exit(1);
	}
	else
	{
		printf("Found %d GPUs!\n", deviceCount);
		gpuSetDevice(0);
		printf("GPU 0 initialized!\n");
	}

	Instance *d_instance;
	Solution *d_solution;
	hiprandState_t* states;
	hipMalloc((void**) &states, nThreads * sizeof(hiprandState_t));


	Instance *inst = loadInstance(fileName);
	//showInstance(inst);
	printf("Load data instance ok!\n");


	Solution *sol = allocationPointersSolution(inst);
	if(fileName[0]=='e'){
		sol = guloso(inst,1,20);
	}else{
		sol = guloso(inst,1,2);
	}
	//showSolution(sol,inst);
	//printf("greedy solution ok!\n");
	size_solution = sizeof(Solution)
							+ sizeof(Ts)*inst->nJobs //vector s
							+ sizeof(TresUsage)*inst->mAgents; //vector resUsage
	

	srand(time(NULL));
	//for(int i=0;i<=10;i++){
	//schc_cpu(sol, inst, 50);
	//}
	//getchar();
	d_instance = createGPUInstance(inst, inst->nJobs, inst->mAgents);
	d_solution = createGPUsolution(sol,inst->nJobs, inst->mAgents);
	unsigned int *h_rank = (unsigned int*)malloc(sizeof(unsigned int)*inst->nJobs*inst->mAgents);
	memset(h_rank,0,sizeof(unsigned int)*inst->nJobs*inst->mAgents);
	unsigned int *d_rank;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	gpuMalloc((void* ) &d_rank, sizeof(unsigned int)*inst->nJobs*inst->mAgents);
	gpuMemcpy(d_rank, h_rank,sizeof(unsigned int)*inst->nJobs*inst->mAgents , hipMemcpyHostToDevice);


	gettimeofday(&inicio, NULL);
	//schc_cpu(sol,inst,100);
	hipEventRecord(start);

	SCHC<<<1,nThreads>>>(d_instance,d_solution, time(NULL),d_rank, states, 100);

	hipEventRecord(stop);

	gpuMemcpy(sol, d_solution, size_solution, hipMemcpyDeviceToHost);
	gpuMemcpy(h_rank, d_rank,sizeof(unsigned int)*inst->nJobs*inst->mAgents , hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("time: %.4fms\n", milliseconds);
	gettimeofday(&fim, NULL);
	tmili = (int) (1000 * (fim.tv_sec - inicio.tv_sec) + (fim.tv_usec - inicio.tv_usec) / 1000);
	printf("tempo: %d\n",tmili);
	//reallocation pointers of Instance
	inst->cost = (Tcost*)(inst+1);
	inst->resourcesAgent =(TresourcesAgent*) (inst->cost +(inst->nJobs*inst->mAgents));
	inst->capacity =(Tcapacity*) (inst->resourcesAgent + (inst->nJobs*inst->mAgents));

	//reallocation pointers of Solution
	sol->s = (Ts*)(sol+1);
	sol->resUsage = (TresUsage*)(sol->s + inst->nJobs);

	showSolution(sol,inst);
	for(i=0;i<inst->nJobs;i++){
		for(j=0;j<inst->mAgents;j++){
			printf("Qnt Job %d foi alocada no Agente %d: %d\n",i+1,j+1,h_rank[i*inst->mAgents+j]);
		}
	}
	createDat(inst, h_rank, fileName);
	gpuFree(d_instance);
	gpuFree(d_solution);
	free(inst);
	free(sol);
	printf("program finished successfully!\n");
	return 0;
}
