#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>


#include "gpulib/types.h"
#include "gpulib/gpu.cuh"

#include "Instance.h"
#include "Solution.h"
#include "gSolution.cuh"
#include "guloso.h"

const int nThreads = 1024;
int main(int argc, char *argv[]){
//int main(){
	struct timeval inicio;
	struct timeval fim;
	int tmili;
	size_t size_solution;
	const char *fileName = argv[1];
	//const char *fileName = "a05100";
	int deviceCount = 0;
	//int i;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(1);
	}
	if(deviceCount == 0)
	{
		printf("No GPU found :(");
		exit(1);
	}
	else
	{
		printf("Found %d GPUs!\n", deviceCount);
		gpuSetDevice(0);
		printf("GPU 0 initialized!\n");
	}

	Instance *d_instance;
	Solution *d_solution;
	hiprandState_t* states;
	hipMalloc((void**) &states, nThreads * sizeof(hiprandState_t));


	Instance *inst = loadInstance(fileName);
	//showInstance(inst);
	printf("Load data instance ok!\n");


	Solution *sol = allocationPointersSolution(inst);
	if(fileName[0]=='e'){
		sol = guloso(inst,1,20);
	}else{
		sol = guloso(inst,1,2);
	}
	//showSolution(sol,inst);
	printf("greedy solution ok!\n");
	size_solution = sizeof(Solution)
							+ sizeof(Ts)*inst->nJobs //vector s
							+ sizeof(TresUsage)*inst->mAgents; //vector resUsage
	

	srand(time(NULL));
	//for(int i=0;i<=10;i++){
	//schc_cpu(sol, inst, 50);
	//}
	//getchar();

	d_instance = createGPUInstance(inst, inst->nJobs, inst->mAgents);
	d_solution = createGPUsolution(sol,inst->nJobs, inst->mAgents);


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	gettimeofday(&inicio, NULL);
	//schc_cpu(sol,inst,100);
	hipEventRecord(start);

	SCHC<<<1,nThreads>>>(d_instance,d_solution, time(NULL), states, 100);

	hipEventRecord(stop);

	gpuMemcpy(sol, d_solution, size_solution, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("time: %.4fms\n", milliseconds);
	gettimeofday(&fim, NULL);
	tmili = (int) (1000 * (fim.tv_sec - inicio.tv_sec) + (fim.tv_usec - inicio.tv_usec) / 1000);
	printf("tempo: %d\n",tmili);
	showSolution(sol,inst);
	gpuFree(d_instance);
	gpuFree(d_solution);
	free(inst);
	free(sol);
	printf("program finished successfully!\n");
	return 0;
}
