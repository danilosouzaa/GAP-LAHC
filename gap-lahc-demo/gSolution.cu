#include "hip/hip_runtime.h"
#include "gSolution.cuh"

const int nThreads = 576;
const int nBlocks = 4;

__global__ void SCHC(Instance *inst, Solution *sol, unsigned int *seed, unsigned int *rank, hiprandState_t* states, int L_c, int max_ite)
{
		// Parameters of SCHC
		int B_c;
		int N_c;

		// Variation in solution
		int delta;

		// Variable auxiliary
		int aux;

		//Solutions Shared
		__shared__ Solution s[nThreads];
		__shared__ int costFinal[nThreads];

		// Minimal cost
		int c_min;

		// Vector with position for ejection chain
		short int pos[8];

		// Option of neighborhood
		short int op;
		// Size of Ejection Chain
		short int size_chain;

		// iterators and auxiliary
		int i,j,k, ite, flag;

		//Allocation of vetor s and resUsage for thread.
		//s[threadIdx.x].costFinal = (TcostFinal*)malloc(sizeof(TcostFinal)*nThreads);
		s[threadIdx.x].s = (Ts*)malloc(sizeof(Ts)*inst->nJobs);
		s[threadIdx.x].resUsage = (TresUsage*)malloc(sizeof(TresUsage)*inst->mAgents);

		//Initialize hiprand 
		hiprand_init(seed[blockIdx.x*nThreads + threadIdx.x],blockIdx.x*nThreads + threadIdx.x,0,&states[blockIdx.x*nThreads + threadIdx.x]);
		
		//Copy solution initial (Solution per block)
		costFinal[threadIdx.x] = sol->costFinal[blockIdx.x];
		for(i=0; i<inst->nJobs; i++)
		{
			s[threadIdx.x].s[i] = sol->s[i + blockIdx.x*inst->nJobs];
		}
		for(i=0; i<inst->mAgents; i++)
		{
			s[threadIdx.x].resUsage[i] = sol->resUsage[i + blockIdx.x*inst->mAgents];
		}
		
		//Define size of L_c for random number between 100 and 150
		L_c = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%101 + 50;

		// Initial parametres
		B_c = costFinal[threadIdx.x];
		N_c = 0;
		ite = 0;
		//Loop with conditional number maximal of iteration
		while(ite<=max_ite)
		{
			// Loop for find a solution feasible
			do
			{	
				//op receive what neighborhood (random 0-1)
				op = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%2;
						
				aux=0; //value 0 for movement infeasible
				delta = 0; //initial value delta with 0
				if(op == 1) //neighborhood 1 change agent
				{
					pos[0] = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%inst->nJobs; //define which job will be modified
					pos[1] = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%inst->mAgents; // define new agent for job
					delta = inst->cost[pos[0]*inst->mAgents + pos[1]] - inst->cost[pos[0]*inst->mAgents + ((int)s[threadIdx.x].s[pos[0]])]; //Calculate the delta value 
					// Conditional of Capacity and Resource Usage
					if(( s[threadIdx.x].resUsage[pos[1]] + inst->resourcesAgent[pos[0]*inst->mAgents + pos[1]] <= inst->capacity[pos[1]])&&
					(s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[pos[0]])] - inst->resourcesAgent[pos[0]*inst->mAgents + ((int)s[threadIdx.x].s[pos[0]])] <= inst->capacity[((int)s[threadIdx.x].s[pos[0]])]))
					{
						aux=1; //value 1 for movement feasible
					}
				}
				else
				{
					aux = 1; //value 1 for movement feasible (assumption)
					size_chain = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%6 + 2; //define size_chain 
					pos[0] = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%inst->nJobs; //first agent define for ejection chain
					delta -= inst->cost[ pos[0]*inst->mAgents+ ((int)s[threadIdx.x].s[pos[0]])]; //Update delta
					pos[size_chain] = inst->nJobs-1;//initialize last position for comparation
					do{
						pos[size_chain]--;
					}while(((int)s[threadIdx.x].s[pos[0]])== ((int)s[threadIdx.x].s[pos[size_chain]]));//verify if first equal the last
					for(i=1; i<=size_chain; i++) //runs ejection chain
					{	
						pos[i] = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%inst->nJobs;//define position i of ejection chain
						k = pos[i]; //k is auxiliary
						//loop for verify feasibly of moviment						
						do{ 
							flag = 0;
							for(j=0; j<i; j++)//verify if the job has already been selected
							{
								if(pos[i]==pos[j])
								{
									flag = 1;  //flag 1 if position has already been selected
									break; //end comparation
								}
							}
							//verify if the position are different, and satisfy the resources and capacity
							if(
								(((int)s[threadIdx.x].s[pos[i]]) != ((int)s[threadIdx.x].s[pos[i-1]]))
								&&(((int)s[threadIdx.x].s[pos[0]]) != ((int)s[threadIdx.x].s[pos[size_chain]])) 
								&&(flag!=1)
								&&(s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[pos[i]])] - inst->resourcesAgent[pos[i]*inst->mAgents + ((int)s[threadIdx.x].s[pos[i]])] + inst->resourcesAgent[pos[i-1]*inst->mAgents + ((int)s[threadIdx.x].s[pos[i]])] <= inst->capacity[((int)s[threadIdx.x].s[pos[i]])])){
								break; //if yes, next position is randomly selected
							}
							pos[i]=(pos[i]+1)%(inst->nJobs);// if no, position is incremment 
						}while(pos[i]!=k);
						if(k==pos[i]){//verify if it was possible get a ejection chain
							aux=0;//if true, no ejection chain was generated
							break;
						}
						delta += inst->cost[pos[i-1]*inst->mAgents+((int)s[threadIdx.x].s[pos[i]])];//update delta 
						delta -= inst->cost[pos[i]*inst->mAgents+((int)s[threadIdx.x].s[pos[i]])];
					}
					delta += inst->cost[pos[size_chain]*inst->mAgents + ((int)s[threadIdx.x].s[pos[0]])];//update with last and first position
					if(s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[pos[0]])] - inst->resourcesAgent[pos[0]*inst->mAgents + ((int)s[threadIdx.x].s[pos[0]])] + inst->resourcesAgent[pos[size_chain]*inst->mAgents + ((int)s[threadIdx.x].s[pos[0]])]>inst->capacity[((int)s[threadIdx.x].s[pos[0]])])
					{
						aux=0;
					}
				}
				
			}
			while(aux==0);
			ite++;
			if ((costFinal[threadIdx.x] + delta < B_c)||(costFinal[threadIdx.x] + delta <= costFinal[threadIdx.x]))
			{
				costFinal[threadIdx.x] += delta;
				if(op==1)
				{
					s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[pos[0]])] -= inst->resourcesAgent[pos[0]*inst->mAgents + ((int)s[threadIdx.x].s[pos[0]]) ];
					s[threadIdx.x].resUsage[pos[1]] += inst->resourcesAgent[pos[0]*inst->mAgents + pos[1]];
					s[threadIdx.x].s[pos[0]] = ((char)pos[1]);
				}
				else
				{
					s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[pos[0]])] += inst->resourcesAgent[pos[size_chain]*inst->mAgents + ((int)s[threadIdx.x].s[pos[0]])];
					s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[pos[0]])] -= inst->resourcesAgent[pos[0]*inst->mAgents + ((int)s[threadIdx.x].s[pos[0]])];
					aux = ((int)s[threadIdx.x].s[pos[0]]);
					for(i=1; i<=size_chain; i++)
					{
						s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[pos[i]])] += inst->resourcesAgent[pos[i-1]*inst->mAgents + ((int)s[threadIdx.x].s[pos[i]])];
						s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[pos[i]])] -= inst->resourcesAgent[pos[i]*inst->mAgents + ((int)s[threadIdx.x].s[pos[i]])];
						s[threadIdx.x].s[pos[i-1]] = s[threadIdx.x].s[pos[i]];
					}
					s[threadIdx.x].s[pos[size_chain]] = ((char)aux);
				}
			}
			N_c++;
			if(N_c >= L_c)
			{
				B_c = costFinal[threadIdx.x];
				N_c = 0;
			}
			
		}
		
		
		for(j=0;j<inst->nJobs;j++){
			atomicInc(&rank[j * inst->mAgents + ((int)s[threadIdx.x].s[j])],(nThreads*nBlocks)+1);
		}


		if(threadIdx.x < 1)
		{
			c_min = costFinal[threadIdx.x];
			aux = 0;
			for(i=1; i<nThreads; i++)
			{	

				if(costFinal[i]<c_min)
				{	
					c_min = costFinal[i];
					aux = i;
				}
			}
			sol->costFinal[blockIdx.x] = costFinal[aux];
			for(j=0; j<inst->nJobs; j++)
			{	
				sol->s[j + blockIdx.x*inst->nJobs] = s[aux].s[j] ;
			}	
			for(j=0; j<inst->mAgents; j++)
			{
				sol->resUsage[j + blockIdx.x*inst->mAgents] = s[aux].resUsage[j];	
				k = s[aux].resUsage[j];
			}

		}
		__syncthreads();
		//free(s[threadIdx.x].costFinal);
		free(s[threadIdx.x].s);
		free(s[threadIdx.x].resUsage);
}


Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents)
{
	//printf("Begin createGpuSolution!\n");

	size_t size_solution = sizeof(Solution)
												   + sizeof(TcostFinal)*nBlocks
												   + sizeof(Ts)*(nJobs*nBlocks) //vector s
												   + sizeof(TresUsage)*(mAgents*nBlocks); // vector resUsage

	Solution *d_sol;
	gpuMalloc((void**)&d_sol, size_solution);
	//printf("malloc solution ok!\n");
	//getchar();
	gpuMemset(d_sol,0,size_solution);
	//printf("memset Solution ok!\n");
	//getchar();
	h_solution->costFinal = (TcostFinal*)(d_sol+1);
	h_solution->s = (Ts*)(h_solution->costFinal + nBlocks);
	h_solution->resUsage = (TresUsage*)(h_solution->s + (nJobs*nBlocks));

	//printf("adjusting solution GPU pointers\n");
	//getchar();

	gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);

	//printf("memcpy Solution ok!\n");
	//getchar();

	return d_sol;

}

