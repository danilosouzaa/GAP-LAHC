#include "hip/hip_runtime.h"
#include "gSolution.cuh"

const int nThreads = 1024;

__global__ void SCHC(Instance *inst, Solution *sol, unsigned int seed, unsigned int *rank, hiprandState_t* states, int L_c)
{
	int B_c;
	int N_c;
	int delta;
	int aux;
	int test_ite = 0 ;

	__shared__ Solution s[nThreads];
	__shared__ int max_ite;
	if(threadIdx.x<1){
		max_ite = 0;
	}

	int c_min;
	short int aux1;
	short int aux2;
	//short int aux3;
	short int aux_p[10];
	short int op;
	short int t;
	int i,j, ite, flag;
	s[threadIdx.x].s = (short int*)malloc(sizeof(short int)*inst->nJobs);
	s[threadIdx.x].resUsage = (short int*)malloc(sizeof(short int)*inst->mAgents);
	hiprand_init(seed,threadIdx.x,0,&states[threadIdx.x]);
	s[threadIdx.x].costFinal = sol->costFinal;
	for(i=0; i<inst->nJobs; i++)
	{
		s[threadIdx.x].s[i] = sol->s[i];
	}
	for(i=0; i<inst->mAgents; i++)
	{
		s[threadIdx.x].resUsage[i] = sol->resUsage[i];
	}

	B_c = sol->costFinal;
	N_c=0;
	ite=0;
	while(ite<=15000)
	{
		do
		{
			op = hiprand(&states[threadIdx.x])%2;
			//printf("custo final temp: %d\n", s[threadIdx.x].costFinal);
			aux=0;
			// op = 1;
			if(op == 1)
			{
				delta=0;
				aux1 = hiprand(&states[threadIdx.x])%inst->nJobs;
				aux2 = hiprand(&states[threadIdx.x])%inst->mAgents;
				delta = inst->cost[aux1*inst->mAgents+aux2] - inst->cost[aux1*inst->mAgents + s[threadIdx.x].s[aux1]];
				if((s[threadIdx.x].resUsage[aux2] + inst->resourcesAgent[aux1*inst->mAgents+aux2] <= inst->capacity[aux2])&&
						(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] - inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1]] <= inst->capacity[s[threadIdx.x].s[aux1]]))
				{
					aux=1;
				}
			}
			else
			{
				delta=0;
				aux = 1;
				t = hiprand(&states[threadIdx.x])%8 + 2;
				aux_p[0] = hiprand(&states[threadIdx.x])%inst->nJobs;
				delta -= inst->cost[ aux_p[0]*inst->mAgents+s[threadIdx.x].s[aux_p[0]]];
				for(i=1; i<=t; i++)
				{

					aux_p[i] = hiprand(&states[threadIdx.x])%inst->nJobs;
					aux1 = aux_p[i];
					do{
						flag = 0;
						for(j=0; j<i; j++)
						{
							if(aux_p[i]==aux_p[j])
							{
								flag = 1;
							}
						}
						if((s[threadIdx.x].s[aux_p[i]] != s[threadIdx.x].s[aux_p[i-1]])&&( s[threadIdx.x].s[aux_p[0]] != s[threadIdx.x].s[aux_p[t]] ) &&(flag!=1)&&(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] - inst->resourcesAgent[aux_p[i]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]] + inst->resourcesAgent[aux_p[i-1]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]] <= inst->capacity[s[threadIdx.x].s[aux_p[i]]])){
							break;
						}
						aux_p[i]=(aux_p[i]+1)%(inst->nJobs);
					}while(aux_p[i]!=aux1);
					if(aux1==aux_p[i]){
						aux=0;
					}


					/*while((i==t)&&(s[threadIdx.x].s[aux_p[0]]==s[threadIdx.x].s[aux_p[t]]))
						{
							aux_p[i] = hiprand(&states[threadIdx.x])%inst->nJobs;
						}
						for(j=0; j<i; j++)
						{
							if(aux_p[i]==aux_p[j])
							{
								flag = 1;
							}
						}*/
					delta += inst->cost[aux_p[i-1]*inst->mAgents+s[threadIdx.x].s[aux_p[i]]];
					delta -= inst->cost[aux_p[i]*inst->mAgents+s[threadIdx.x].s[aux_p[i]]];


				}
				delta += inst->cost[aux_p[t]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]];
				if(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[0]]] - inst->resourcesAgent[aux_p[0]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]] + inst->resourcesAgent[aux_p[t]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]]>inst->capacity[s[threadIdx.x].s[aux_p[0]]])
				{
					aux=0;
				}
			}
			test_ite++;
		}
		while(aux==0);
		if(test_ite> max_ite){
			max_ite = test_ite;
		}
		test_ite = 0;
		if ((s[threadIdx.x].costFinal + delta < B_c)||(s[threadIdx.x].costFinal + delta <= s[threadIdx.x].costFinal))
		{
			s[threadIdx.x].costFinal += delta;
			if(op==1)
			{
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] -= inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1] ];
				s[threadIdx.x].resUsage[aux2] += inst->resourcesAgent[aux1*inst->mAgents + aux2];
				s[threadIdx.x].s[aux1] = aux2;
			}
			else
			{
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[0]]] += inst->resourcesAgent[aux_p[t]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]];
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[0]]] -= inst->resourcesAgent[aux_p[0]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]];
				aux = s[threadIdx.x].s[aux_p[0]];
				for(i=1; i<=t; i++)
				{
					s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] += inst->resourcesAgent[aux_p[i-1]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]];
					s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] -= inst->resourcesAgent[aux_p[i]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]];
					s[threadIdx.x].s[aux_p[i-1]] = s[threadIdx.x].s[aux_p[i]];
				}
				s[threadIdx.x].s[aux_p[t]] = aux;
			}
		}
		N_c++;
		if(N_c >= L_c)
		{
			B_c = s[threadIdx.x].costFinal;
			N_c = 0;
		}
		ite++;
	}

	if(threadIdx.x < 1)
	{
		c_min = s[threadIdx.x].costFinal;
		for(i=0; i<nThreads; i++)
		{
			for(j=0;j<inst->nJobs;j++){
				atomicInc(&rank[j * inst->mAgents +s[i].s[j]],nThreads+1);
			}

			if(s[i].costFinal<c_min)
			{
				c_min = s[i].costFinal;
				sol->costFinal = s[i].costFinal;
				for(j=0; j<inst->nJobs; j++)
				{
					sol->s[j] = s[i].s[j] ;
				}
				for(j=0; j<inst->mAgents; j++)
				{
					sol->resUsage[j] = s[i].resUsage[j];
				}
			}
		}
	}

	free(s[threadIdx.x].s);
	free(s[threadIdx.x].resUsage);
	if(threadIdx.x <1 )
	{
		printf("\n%d ---- ", c_min);
		printf("%d ----", max_ite);
	}

}


Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents)
{
	printf("Begin createGpuSolution!\n");

	size_t size_solution = sizeof(Solution)
                        				   + sizeof(Ts)*nJobs //vector s
                        				   + sizeof(TresUsage)*mAgents; // vector resUsage
	Solution *d_sol;
	gpuMalloc((void**)&d_sol, size_solution);
	printf("malloc solution ok!\n");
	//getchar();
	gpuMemset(d_sol,0,size_solution);
	printf("memset Solution ok!\n");
	//getchar();

	h_solution->s = (Ts*)(d_sol+1);
	h_solution->resUsage = (TresUsage*)(h_solution->s + nJobs);

	printf("adjusting solution GPU pointers\n");
	//getchar();

	gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);

	printf("memcpy Solution ok!\n");
	//getchar();

	return d_sol;

}

