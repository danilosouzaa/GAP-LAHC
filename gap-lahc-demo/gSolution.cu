#include "gSolution.cuh"

Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents){
	printf("Begin createGpuSolution!\n");

	size_solution = sizeof(Solution)
						+ sizeof(Ts)*nJobs //vector s
						+ sizeof(TresUsage)*mAgents; // vector resUsage
	Solution *d_sol;
	gpuMalloc((void**)&d_sol, size_solution);
	printf("malloc solution ok!");
	getchar();
	gpuMemset(d_sol,0,size_solution);
	printf("memset Solution ok!")
	getchar();

	h_solution->s = (Ts*)(d_sol+1);
	h_solution->resUsage = (TresUsage*)(h_solution->s + nJobs);

	printf("adjusting solution GPU pointers");
	getchar();

	gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);

	printf("memcpy Solution ok!");
	getchar();

	return d_sol;

}
