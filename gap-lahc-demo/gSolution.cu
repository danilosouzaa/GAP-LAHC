#include "hip/hip_runtime.h"
#include "gSolution.cuh"

const int nThreads =1024;

__global__ void SCHC(Instance *inst, Solution *sol, unsigned int seed, hiprandState_t* states, int L_c)
{
    int B_c;
    int N_c;
    int delta;
    int aux;
    __shared__ Solution s[nThreads];
    int c_min;
    short int aux1;
    short int aux2;
    short int aux3;
    short int aux_p[10];
    short int op;
    short int t;
    int i;
    s[threadIdx.x].s = (short int*)malloc(sizeof(short int)*inst->nJobs);
    s[threadIdx.x].resUsage = (short int*)malloc(sizeof(short int)*inst->mAgents);
    hiprand_init(seed,threadIdx.x,0,&states[threadIdx.x]);
    s[threadIdx.x].costFinal = sol->costFinal;
    for(i=0; i<inst->nJobs; i++)
    {
        s[threadIdx.x].s[i] = sol->s[i];
    }
    for(i=0; i<inst->mAgents; i++)
    {
        s[threadIdx.x].resUsage[i] = sol->resUsage[i];
    }

    B_c = sol->costFinal;
    N_c=0;
    i=0;
    while(i<=300000)
    {
        do
        {
            op = hiprand(&states[threadIdx.x])%2;
            //printf("custo final temp: %d\n", s[threadIdx.x].costFinal);
            aux=0;
            //op = 0;
            if(op == 1)
            {
                delta=0;
                aux1 = hiprand(&states[threadIdx.x])%inst->nJobs;
                aux2 = hiprand(&states[threadIdx.x])%inst->mAgents;
                delta = inst->cost[aux1*inst->mAgents+aux2] - inst->cost[aux1*inst->mAgents + s[threadIdx.x].s[aux1]];
                if((s[threadIdx.x].resUsage[aux2] + inst->resourcesAgent[aux1*inst->mAgents+aux2] <= inst->capacity[aux2])&&
                        (s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] - inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1]] <= inst->capacity[s[threadIdx.x].s[aux1]]))
                {
                    aux=1;
                }
            }
            else
            {
                delta=0;
                aux = 1;
                t = hiprand(&states[threadIdx.x])%8 + 2;
                aux_p[0] = hiprand(&states[threadIdx.x])%inst->nJobs;
                delta -= inst->cost[ aux_p[0]*inst->mAgents+s[threadIdx.x].s[aux_p[0]]];
                for(i=1; i<=t; i++)
                {
                    do
                    {
                        aux_p[i] = hiprand(&states[threadIdx.x])%inst->nJobs;
                        while((t==i)&&(s[threadIdx.x].s[aux_p[i]]==s[threadIdx.x].s[aux_p[0]]))
                        {
                            aux_p[i] = hiprand(&states[threadIdx.x])%inst->nJobs;
                        }
                    }
                    while(s[threadIdx.x].s[aux_p[i]]==s[threadIdx.x].s[aux_p[i-1]]);
                    delta += inst->cost[aux_p[i-1]*inst->mAgents+s[threadIdx.x].s[aux_p[i]]];
                    delta -= inst->cost[aux_p[i]*inst->mAgents+s[threadIdx.x].s[aux_p[i]]];
                    if(t==i)
                    {
                        delta += inst->cost[aux_p[i]*inst->mAgents+s[threadIdx.x].s[aux_p[0]]];
                        if(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[0]]] - inst->resourcesAgent[aux_p[0]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]] + inst->resourcesAgent[aux_p[i]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]]>inst->capacity[s[threadIdx.x].s[aux_p[0]]])
                        {
                            aux=0;
                            break;
                        }

                    }
                    else
                    {

                        if(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] - inst->resourcesAgent[aux_p[i]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]] + inst->resourcesAgent[aux_p[i-1]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]]>inst->capacity[s[threadIdx.x].s[aux_p[i]]])
                        {
                            aux=0;
                            break;
                        }
                    }
                }
            }
        }
        while(aux==0);

        if ((s[threadIdx.x].costFinal + delta < B_c)||(s[threadIdx.x].costFinal + delta <= s[threadIdx.x].costFinal))
        {
            s[threadIdx.x].costFinal += delta;
            if(op==1)
            {
                s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] -= inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1] ];
                s[threadIdx.x].resUsage[aux2] += inst->resourcesAgent[aux1*inst->mAgents + aux2];
                s[threadIdx.x].s[aux1] = aux2;
            }
            else
            {
                s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[0]]] += inst->resourcesAgent[aux_p[t]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]];
                s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[0]]] -= inst->resourcesAgent[aux_p[0]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]];
                aux = s[threadIdx.x].s[aux_p[0]];
                for(i=1; i<=t; i++)
                {
                    s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] += inst->resourcesAgent[aux_p[i-1]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]];
                    s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] -= inst->resourcesAgent[aux_p[i]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]];
                    s[threadIdx.x].s[aux_p[i-1]] = s[threadIdx.x].s[aux_p[i]];
                }
                s[threadIdx.x].s[aux_p[t]] = aux;
            }
        }
        N_c++;
        if(N_c >= L_c)
        {
            B_c = s[threadIdx.x].costFinal;
            N_c = 0;
        }
        i++;
    }

    if(threadIdx.x < 1)
    {
        c_min = s[threadIdx.x].costFinal;
        for(i=0; i<nThreads; i++)
        {
            if(s[i].costFinal<c_min)
            {
                c_min = s[i].costFinal;
            }
        }
    }

    free(s[threadIdx.x].s);
    free(s[threadIdx.x].resUsage);

    if(threadIdx.x <1 )
    {
        printf("\ntestes: %d\n", c_min);
    }
}


Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents)
{
    printf("Begin createGpuSolution!\n");

    size_t size_solution = sizeof(Solution)
                           + sizeof(Ts)*nJobs //vector s
                           + sizeof(TresUsage)*mAgents; // vector resUsage
    Solution *d_sol;
    gpuMalloc((void**)&d_sol, size_solution);
    printf("malloc solution ok!\n");
    //getchar();
    gpuMemset(d_sol,0,size_solution);
    printf("memset Solution ok!\n");
    //getchar();

    h_solution->s = (Ts*)(d_sol+1);
    h_solution->resUsage = (TresUsage*)(h_solution->s + nJobs);

    printf("adjusting solution GPU pointers\n");
    //getchar();

    gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);

    printf("memcpy Solution ok!\n");
    //getchar();

    return d_sol;

}

