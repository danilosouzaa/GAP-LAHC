#include "hip/hip_runtime.h"
#include "gSolution.cuh"

const int nThreads =1024;

__global__ void SCHC(Instance *inst, Solution *sol, unsigned int seed, hiprandState_t* states, int L_c){
	int B_c;
	int N_c;
	int delta;
	int aux;
	__shared__ Solution s[nThreads];
	int c_min;
	short int aux1;
	short int aux2;
	short int aux3;
	short int op;
	int i;
	s[threadIdx.x].s = (short int*)malloc(sizeof(short int)*inst->nJobs);
	s[threadIdx.x].resUsage = (short int*)malloc(sizeof(short int)*inst->mAgents);
	hiprand_init(seed,threadIdx.x,0,&states[threadIdx.x]);
	s[threadIdx.x].costFinal = sol->costFinal;
	for(i=0;i<inst->nJobs;i++){
		s[threadIdx.x].s[i] = sol->s[i];
	}
	for(i=0;i<inst->mAgents;i++){
		s[threadIdx.x].resUsage[i] = sol->resUsage[i];
	}

	B_c = sol->costFinal;
	N_c=0;
	i=0;
	while(i<=60000){

		op = hiprand(&states[threadIdx.x])%2;
		//printf("custo final temp: %d\n", s[threadIdx.x].costFinal);
		
		//op = 0;
		if(op == 1){
			do{	delta=0;
				aux1 = hiprand(&states[threadIdx.x])%inst->nJobs;
				aux2 = hiprand(&states[threadIdx.x])%inst->mAgents;
				delta = inst->cost[aux1*inst->mAgents+aux2] - inst->cost[aux1*inst->mAgents + s[threadIdx.x].s[aux1]];
			}while((s[threadIdx.x].resUsage[aux2] + inst->resourcesAgent[aux1*inst->mAgents+aux2] > inst->capacity[aux2])||
					(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] - inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1]] > inst->capacity[s[threadIdx.x].s[aux1]]));
		}else{	
			do{
			delta=0;
			do{
				aux1 = hiprand(&states[threadIdx.x])%inst->nJobs;
				aux2 = hiprand(&states[threadIdx.x])%inst->nJobs;
				aux3 = hiprand(&states[threadIdx.x])%inst->nJobs;
			}while((s[threadIdx.x].s[aux1]==s[threadIdx.x].s[aux2])||(s[threadIdx.x].s[aux2]==s[threadIdx.x].s[aux3])||(s[threadIdx.x].s[aux1]==s[threadIdx.x].s[aux3]));
			delta = inst->cost[aux1*inst->mAgents+s[threadIdx.x].s[aux2]] + inst->cost[aux2*inst->mAgents+s[threadIdx.x].s[aux3]] + inst->cost[aux3*inst->mAgents+s[threadIdx.x].s[aux1]]  - inst->cost[aux1*inst->mAgents+s[threadIdx.x].s[aux1]] - inst->cost[aux2*inst->mAgents+s[threadIdx.x].s[aux2]] - inst->cost[aux3*inst->mAgents+s[threadIdx.x].s[aux3]];
			}while(
(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] - inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1]] + inst->resourcesAgent[aux3*inst->mAgents + s[threadIdx.x].s[aux1]]>inst->capacity[s[threadIdx.x].s[aux1]])
||
(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux2]] - inst->resourcesAgent[aux2*inst->mAgents + s[threadIdx.x].s[aux2]] + inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux2]]>inst->capacity[s[threadIdx.x].s[aux2]])
||
(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux3]] - inst->resourcesAgent[aux3*inst->mAgents + s[threadIdx.x].s[aux3]] + inst->resourcesAgent[aux2*inst->mAgents + s[threadIdx.x].s[aux3]]>inst->capacity[s[threadIdx.x].s[aux3]])
 );
		}

		if ((s[threadIdx.x].costFinal + delta < B_c)||(s[threadIdx.x].costFinal + delta <= s[threadIdx.x].costFinal)){
			s[threadIdx.x].costFinal += delta;
			if(op==1){
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] -= inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1] ];
				s[threadIdx.x].resUsage[aux2] += inst->resourcesAgent[aux1*inst->mAgents + aux2];
				s[threadIdx.x].s[aux1] = aux2;
			}else{
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux2]] += inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux2]];
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux2]] -= inst->resourcesAgent[aux2*inst->mAgents + s[threadIdx.x].s[aux2]];
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux3]] += inst->resourcesAgent[aux2*inst->mAgents + s[threadIdx.x].s[aux3]];
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux3]] -= inst->resourcesAgent[aux3*inst->mAgents + s[threadIdx.x].s[aux3]];
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] += inst->resourcesAgent[aux3*inst->mAgents + s[threadIdx.x].s[aux1]]; 
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] -= inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1]];
				aux = s[threadIdx.x].s[aux1];
				s[threadIdx.x].s[aux1] = s[threadIdx.x].s[aux2];
				s[threadIdx.x].s[aux2] = s[threadIdx.x].s[aux3];
				s[threadIdx.x].s[aux3] = aux;
			}
		}
		N_c++;
		if(N_c >= L_c){
			B_c = s[threadIdx.x].costFinal;
			N_c = 0;
		}
		i++;
	}
	
	if(threadIdx.x < 1){
		c_min = s[threadIdx.x].costFinal;
		for(i=0;i<nThreads;i++){
			if(s[i].costFinal<c_min){
				c_min = s[i].costFinal;	
			}	  
		}	
	}
	
	free(s[threadIdx.x].s);
	free(s[threadIdx.x].resUsage);
	
	if(threadIdx.x <1 ){
		printf("\ntestes: %d\n", c_min);
	}
}


Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents){
	printf("Begin createGpuSolution!\n");

	size_t size_solution = sizeof(Solution)
										+ sizeof(Ts)*nJobs //vector s
										+ sizeof(TresUsage)*mAgents; // vector resUsage
	Solution *d_sol;
	gpuMalloc((void**)&d_sol, size_solution);
	printf("malloc solution ok!\n");
	//getchar();
	gpuMemset(d_sol,0,size_solution);
	printf("memset Solution ok!\n");
	//getchar();

	h_solution->s = (Ts*)(d_sol+1);
	h_solution->resUsage = (TresUsage*)(h_solution->s + nJobs);

	printf("adjusting solution GPU pointers\n");
	//getchar();

	gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);

	printf("memcpy Solution ok!\n");
	//getchar();

	return d_sol;

}
