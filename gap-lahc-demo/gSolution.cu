#include "hip/hip_runtime.h"
#include "gSolution.cuh"


__global__ void SCHC(Instance *inst, Solution *sol, unsigned int seed, hiprandState_t* states, int L_c){
	int B_c;
	int N_c;
	int delta;
	__shared__ Solution s[20];
	short int aux1;
	short int aux2;
	short int op;
	int i;
	hiprand_init(seed,threadIdx.x,0,&states[threadIdx.x]);

	s[threadIdx.x].s = (short int*)malloc(sizeof(short int)*inst->nJobs);
	s[threadIdx.x].resUsage = (short int*)malloc(sizeof(short int)*inst->mAgents);

	s[threadIdx.x].costFinal = sol->costFinal;
	for(i=0;i<inst->nJobs;i++){
		s[threadIdx.x].s[i] = sol->s[i];
	}
	for(i=0;i<inst->mAgents;i++){
		s[threadIdx.x].resUsage[i] = sol->resUsage[i];
	}

	B_c = sol->costFinal;
	N_c=0;
	i=0;
	while(i<=100000){
		op = hiprand(&states[threadIdx.x])%2;
		//printf("custo final temp: %d\n", s[threadIdx.x].costFinal);
		//op = 1;
		if(op == 1){
			do{
				aux1 = hiprand(&states[threadIdx.x])%inst->nJobs;
				aux2 = hiprand(&states[threadIdx.x])%inst->mAgents;
				//printf("teste1: %d\n", inst->cost[aux1[threadIdx.x]*inst->mAgents+aux2[threadIdx.x]]);
				//printf("teste2: %d\n", inst->cost[aux1[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux1[threadIdx.x]]]);
				delta = inst->cost[aux1*inst->mAgents+aux2] - inst->cost[aux1*inst->mAgents + s[threadIdx.x].s[aux1]];
			}while((s[threadIdx.x].resUsage[aux2] + inst->resourcesAgent[aux1*inst->mAgents+aux2] > inst->capacity[aux2])||
					(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] - inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1]] > inst->capacity[s[threadIdx.x].s[aux1]]));
		}else{
			do{
				aux1 = hiprand(&states[threadIdx.x])%inst->nJobs;
				do{
					aux2 = hiprand(&states[threadIdx.x])%inst->nJobs;
				}while(aux1==aux2);
				//printf("aux1: %d aux2: %d \n", aux1,aux2);
				delta =  inst->cost[aux1*inst->mAgents + s[threadIdx.x].s[aux2]] + inst->cost[aux2*inst->mAgents + s[threadIdx.x].s[aux1]]- inst->cost[aux1*inst->mAgents + s[threadIdx.x].s[aux1]] - inst->cost[aux2*inst->mAgents + s[threadIdx.x].s[aux2]];
			}while((s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] - inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1]] + inst->resourcesAgent[aux2*inst->mAgents + s[threadIdx.x].s[aux1]]>inst->capacity[s[threadIdx.x].s[aux1]])
					||(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux2]] - inst->resourcesAgent[aux2*inst->mAgents + s[threadIdx.x].s[aux2]] +  inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux2]]> inst->capacity[s[threadIdx.x].s[aux2]]));
		}
		//printf("Delta: %d i: %d\n", delta[threadIdx.x], i[threadIdx.x]);
		if ((s[threadIdx.x].costFinal + delta < B_c)||(s[threadIdx.x].costFinal+delta <= s[threadIdx.x].costFinal)){
			s[threadIdx.x].costFinal += delta;
			if(op==1){
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] -= inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1] ];
				s[threadIdx.x].resUsage[aux2] += inst->resourcesAgent[aux1*inst->mAgents + aux2];
				s[threadIdx.x].s[aux1] = aux2;
			}else{
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]]-= inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1]];
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]]+= inst->resourcesAgent[aux2*inst->mAgents + s[threadIdx.x].s[aux1]];
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux2]]-= inst->resourcesAgent[aux2*inst->mAgents + s[threadIdx.x].s[aux2]];
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux2]]+= inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux2]];
				delta = s[threadIdx.x].s[aux1];
				s[threadIdx.x].s[aux1] = s[threadIdx.x].s[aux2];
				s[threadIdx.x].s[aux2] = delta;
			}
		}
		N_c++;
		if(N_c >= L_c){
			B_c = s[threadIdx.x].costFinal;
			N_c = 0;
		}
		i++;
	}

	printf("Custo final: %d\n", s[threadIdx.x].costFinal);

}


Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents){
	printf("Begin createGpuSolution!\n");

	size_t size_solution = sizeof(Solution)
										+ sizeof(Ts)*nJobs //vector s
										+ sizeof(TresUsage)*mAgents; // vector resUsage
	Solution *d_sol;
	gpuMalloc((void**)&d_sol, size_solution);
	printf("malloc solution ok!");
	getchar();
	gpuMemset(d_sol,0,size_solution);
	printf("memset Solution ok!");
	getchar();

	h_solution->s = (Ts*)(d_sol+1);
	h_solution->resUsage = (TresUsage*)(h_solution->s + nJobs);

	printf("adjusting solution GPU pointers");
	getchar();

	gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);

	printf("memcpy Solution ok!");
	getchar();

	return d_sol;

}
