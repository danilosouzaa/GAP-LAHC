#include "hip/hip_runtime.h"
#include "gSolution.cuh"


__global__ void SCHC(Instance *inst, Solution *sol, unsigned int seed, hiprandState_t* states, int L_c){
	int B_c[10];
	int N_c[10];
	int delta[10];
	Solution s[10];
	int aux1[10];
	int aux2[10];
	int op[10];
	int i[10];
	hiprand_init(seed,threadIdx.x,0,&states[threadIdx.x]);

	if(threadIdx.x < 10){
		s[threadIdx.x] = *sol;
		B_c[threadIdx.x] = sol->costFinal;
		N_c[threadIdx.x] = 0;
		i[threadIdx.x]=0;
		while(i[threadIdx.x]<=10000){
			//op[threadIdx.x] = hiprand(&states[threadIdx.x])%2;
			op[threadIdx.x] = 0;
			if(op[threadIdx.x] == 1){
				do{
					aux1[threadIdx.x] = hiprand(&states[threadIdx.x])%inst->nJobs;
					aux2[threadIdx.x] = hiprand(&states[threadIdx.x])%inst->mAgents;
					delta[threadIdx.x] = s[threadIdx.x].costFinal - inst->cost[aux1[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux1[threadIdx.x]]] + inst->cost[aux1[threadIdx.x]*inst->mAgents+aux2[threadIdx.x]];
				}while(s[threadIdx.x].resUsage[aux2[threadIdx.x]] + inst->resourcesAgent[aux1[threadIdx.x]*inst->mAgents+aux2[threadIdx.x]] > inst->capacity[aux2[threadIdx.x]]);
			}else{
				do{
					aux1[threadIdx.x] = hiprand(&states[threadIdx.x])%inst->nJobs;
					do{
						aux2[threadIdx.x] = hiprand(&states[threadIdx.x])%inst->nJobs;
					}while(aux1[threadIdx.x]==aux2[threadIdx.x]);
					delta[threadIdx.x] = s[threadIdx.x].costFinal - inst->cost[aux1[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux1[threadIdx.x]]] - inst->cost[aux2[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux2[threadIdx.x]]];
					delta[threadIdx.x] += inst->cost[aux1[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux2[threadIdx.x]]] + inst->cost[aux2[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux1[threadIdx.x]]];
				}while((s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1[threadIdx.x]]] - inst->resourcesAgent[aux1[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux1[threadIdx.x]]] + inst->resourcesAgent[aux2[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux1[threadIdx.x]]]>inst->capacity[s[threadIdx.x].s[aux1[threadIdx.x]]])
						||(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux2[threadIdx.x]]] - inst->resourcesAgent[aux2[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux2[threadIdx.x]]] +  inst->resourcesAgent[aux1[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux2[threadIdx.x]]]> inst->capacity[s[threadIdx.x].s[aux2[threadIdx.x]]]));
			}
			printf("Delta: %d i: %d\n", delta[threadIdx.x], i[threadIdx.x]);
			if ((delta[threadIdx.x] < B_c[threadIdx.x])||(delta[threadIdx.x]<=s[threadIdx.x].costFinal)){
				s[threadIdx.x].costFinal = delta[threadIdx.x];
				if(op[threadIdx.x]==1){
					s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1[threadIdx.x]]] -= inst->resourcesAgent[aux1[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux1[threadIdx.x]] ];
					s[threadIdx.x].resUsage[aux2[threadIdx.x]] += inst->resourcesAgent[aux1[threadIdx.x]*inst->mAgents + aux2[threadIdx.x]];
					s[threadIdx.x].s[aux1[threadIdx.x]] = aux2[threadIdx.x];
				}else{
					s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1[threadIdx.x]]]-= inst->resourcesAgent[aux1[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux1[threadIdx.x]]];
					s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1[threadIdx.x]]]+= inst->resourcesAgent[aux2[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux1[threadIdx.x]]];
					s[threadIdx.x].resUsage[s[threadIdx.x].s[aux2[threadIdx.x]]]-= inst->resourcesAgent[aux2[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux2[threadIdx.x]]];
					s[threadIdx.x].resUsage[s[threadIdx.x].s[aux2[threadIdx.x]]]+= inst->resourcesAgent[aux1[threadIdx.x]*inst->mAgents + s[threadIdx.x].s[aux2[threadIdx.x]]];
					delta[threadIdx.x] = s[threadIdx.x].s[aux1[threadIdx.x]];
					s[threadIdx.x].s[aux1[threadIdx.x]] = s[threadIdx.x].s[aux2[threadIdx.x]];
					s[threadIdx.x].s[aux2[threadIdx.x]] = delta[threadIdx.x];
				}
			}
			N_c[threadIdx.x]++;
			if(N_c[threadIdx.x] >= L_c){
				B_c[threadIdx.x] = s[threadIdx.x].costFinal;
				N_c[threadIdx.x]=0;
			}


			i[threadIdx.x]++;
		}

		printf("Custo final: %d\n", s[threadIdx.x].costFinal);
	}
}


Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents){
	printf("Begin createGpuSolution!\n");

	size_t size_solution = sizeof(Solution)
								+ sizeof(Ts)*nJobs //vector s
								+ sizeof(TresUsage)*mAgents; // vector resUsage
	Solution *d_sol;
	gpuMalloc((void**)&d_sol, size_solution);
	printf("malloc solution ok!");
	getchar();
	gpuMemset(d_sol,0,size_solution);
	printf("memset Solution ok!");
	getchar();

	h_solution->s = (Ts*)(d_sol+1);
	h_solution->resUsage = (TresUsage*)(h_solution->s + nJobs);

	printf("adjusting solution GPU pointers");
	getchar();

	gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);

	printf("memcpy Solution ok!");
	getchar();

	return d_sol;

}
