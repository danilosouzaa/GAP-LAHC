#include "hip/hip_runtime.h"
#include "gSolution.cuh"

const int nThreads = 576;
const int nBlocks = 28;

__global__ void SCHC(Instance *inst, Solution *sol, unsigned int *seed, unsigned int *rank, hiprandState_t* states, int L_c)
{
		int B_c;
		int N_c;
		int delta;
		int aux, ite_min;
		__shared__ Solution s[nThreads];
		__shared__ int costFinal[nThreads];
		int c_min;
		short int aux_p[8];
		short int op;
		short int t;
		int i,j,k, ite, flag;
		
		s[threadIdx.x].s = (Ts*)malloc(sizeof(Ts)*inst->nJobs);
		s[threadIdx.x].resUsage = (TresUsage*)malloc(sizeof(TresUsage)*inst->mAgents);
		hiprand_init(seed[blockIdx.x*nThreads + threadIdx.x],blockIdx.x*nThreads + threadIdx.x,0,&states[blockIdx.x*nThreads + threadIdx.x]);
		
		costFinal[threadIdx.x] = sol->costFinal[blockIdx.x];
		
		for(i=0; i<inst->nJobs; i++)
		{
			s[threadIdx.x].s[i] = sol->s[i + blockIdx.x*inst->nJobs];
		}
		
		for(i=0; i<inst->mAgents; i++)
		{
			s[threadIdx.x].resUsage[i] = sol->resUsage[i + blockIdx.x*inst->mAgents];
		}
		
		L_c = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%101 + 50;
		B_c = costFinal[threadIdx.x];
		N_c = 0;
		ite = 0;
		ite_min = 0;
		while(ite<=100)
		{
			do
			{
				op = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%2;
				aux=0;
				if(op == 1)
				{
					delta=0;
					aux_p[0] = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%inst->nJobs;
					aux_p[1] = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%inst->mAgents;
					delta = inst->cost[aux_p[0]*inst->mAgents + aux_p[1]] - inst->cost[aux_p[0]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[0]])];
					if(( s[threadIdx.x].resUsage[aux_p[1]] + inst->resourcesAgent[aux_p[0]*inst->mAgents + aux_p[1]] <= inst->capacity[aux_p[1]])&&
							(s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[aux_p[0]])] - inst->resourcesAgent[aux_p[0]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[0]])] <= inst->capacity[((int)s[threadIdx.x].s[aux_p[0]])]))
					{
						aux=1;
					}
				}
				else
				{
					delta=0;
					aux = 1;
					t = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%6 + 2;
					aux_p[0] = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%inst->nJobs;
					delta -= inst->cost[ aux_p[0]*inst->mAgents+ ((int)s[threadIdx.x].s[aux_p[0]])];
					for(i=1; i<=t; i++)
					{
						aux_p[t] = inst->nJobs-1;
						aux_p[i] = hiprand(&states[blockIdx.x*nThreads + threadIdx.x])%inst->nJobs;
						k = aux_p[i];
						do{
							flag = 0;
							for(j=0; j<i; j++)
							{
								if(aux_p[i]==aux_p[j])
								{
									flag = 1;
								}
							}
							if((((int)s[threadIdx.x].s[aux_p[i]]) != ((int)s[threadIdx.x].s[aux_p[i-1]]))&&( ((int)s[threadIdx.x].s[aux_p[0]]) != ((int)s[threadIdx.x].s[aux_p[t]]) ) &&(flag!=1)&&(s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[aux_p[i]])] - inst->resourcesAgent[aux_p[i]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[i]])] + inst->resourcesAgent[aux_p[i-1]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[i]])] <= inst->capacity[((int)s[threadIdx.x].s[aux_p[i]])])){
								break;
							}
							aux_p[i]=(aux_p[i]+1)%(inst->nJobs);
						}while(aux_p[i]!=k);
						if(k==aux_p[i]){
							aux=0;
							break;
						}
						delta += inst->cost[aux_p[i-1]*inst->mAgents+((int)s[threadIdx.x].s[aux_p[i]])];
						delta -= inst->cost[aux_p[i]*inst->mAgents+((int)s[threadIdx.x].s[aux_p[i]])];
					}
					delta += inst->cost[aux_p[t]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[0]])];
					if(s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[aux_p[0]])] - inst->resourcesAgent[aux_p[0]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[0]])] + inst->resourcesAgent[aux_p[t]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[0]])]>inst->capacity[((int)s[threadIdx.x].s[aux_p[0]])])
					{
						aux=0;
					}
				}
				
			}
			while(aux==0);
			ite++;
			if ((costFinal[threadIdx.x] + delta < B_c)||(costFinal[threadIdx.x] + delta <= costFinal[threadIdx.x]))
			{
				costFinal[threadIdx.x] += delta;
				if((costFinal[threadIdx.x]<1714)&&(ite_min==0)){
					ite_min = ite;
					printf("ite min: %d\n",ite_min);
				}
				if(op==1)
				{
					s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[aux_p[0]])] -= inst->resourcesAgent[aux_p[0]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[0]]) ];
					s[threadIdx.x].resUsage[aux_p[1]] += inst->resourcesAgent[aux_p[0]*inst->mAgents + aux_p[1]];
					s[threadIdx.x].s[aux_p[0]] = ((char)aux_p[1]);
				}
				else
				{
					s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[aux_p[0]])] += inst->resourcesAgent[aux_p[t]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[0]])];
					s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[aux_p[0]])] -= inst->resourcesAgent[aux_p[0]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[0]])];
					aux = ((int)s[threadIdx.x].s[aux_p[0]]);
					for(i=1; i<=t; i++)
					{
						s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[aux_p[i]])] += inst->resourcesAgent[aux_p[i-1]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[i]])];
						s[threadIdx.x].resUsage[((int)s[threadIdx.x].s[aux_p[i]])] -= inst->resourcesAgent[aux_p[i]*inst->mAgents + ((int)s[threadIdx.x].s[aux_p[i]])];
						s[threadIdx.x].s[aux_p[i-1]] = s[threadIdx.x].s[aux_p[i]];
					}
					s[threadIdx.x].s[aux_p[t]] = ((char)aux);
				}
			}
			N_c++;
			if(N_c >= L_c)
			{
				B_c = costFinal[threadIdx.x];
				N_c = 0;
			}
			
		}
		
		__syncthreads();
		for(j=0;j<inst->nJobs;j++){
			atomicInc(&rank[j * inst->mAgents + ((int)s[threadIdx.x].s[j])],(nThreads*nBlocks)+1);
		}
		__syncthreads();

		if(threadIdx.x < 1)
		{
			c_min = costFinal[threadIdx.x];
			for(i=1; i<nThreads; i++)
			{	
				
				if(costFinal[threadIdx.x]<c_min)
				{
					c_min = costFinal[threadIdx.x];
					aux = i;
				}
			}
			
			sol->costFinal[blockIdx.x] = costFinal[aux];
			for(j=0; j<inst->nJobs; j++)
			{
				sol->s[j + blockIdx.x*inst->nJobs] = s[aux].s[j] ;
			}
			for(j=0; j<inst->mAgents; j++)
			{
				sol->resUsage[j + blockIdx.x*inst->mAgents] = s[aux].resUsage[j];	
			}
		}	
		//free(s[threadIdx.x].costFinal);
		free(s[threadIdx.x].s);
		free(s[threadIdx.x].resUsage);
}


Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents)
{
	//printf("Begin createGpuSolution!\n");

	size_t size_solution = sizeof(Solution)
												   + sizeof(TcostFinal)*nBlocks
												   + sizeof(Ts)*(nJobs*nBlocks) //vector s
												   + sizeof(TresUsage)*(mAgents*nBlocks); // vector resUsage

	Solution *d_sol;
	gpuMalloc((void**)&d_sol, size_solution);
	//printf("malloc solution ok!\n");
	//getchar();
	gpuMemset(d_sol,0,size_solution);
	//printf("memset Solution ok!\n");
	//getchar();
	h_solution->costFinal = (TcostFinal*)(d_sol+1);
	h_solution->s = (Ts*)(h_solution->costFinal + nBlocks);
	h_solution->resUsage = (TresUsage*)(h_solution->s + (nJobs*nBlocks));

	//printf("adjusting solution GPU pointers\n");
	//getchar();

	gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);

	//printf("memcpy Solution ok!\n");
	//getchar();

	return d_sol;

}

