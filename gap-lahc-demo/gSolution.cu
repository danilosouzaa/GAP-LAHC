#include "hip/hip_runtime.h"
#include "gSolution.cuh"

const int nThreads = 1024;

__global__ void SCHC(Instance *inst, Solution *sol, unsigned int *seed, unsigned int *rank, hiprandState_t* states, int L_c)
{
	int B_c;
	int N_c;
	int delta;
	int aux;
	int test_ite = 0 ;

	__shared__ Solution s[nThreads];
	__shared__ int max_ite;
	if(threadIdx.x<1){
		max_ite = 0;
	}

	long int c_min;
	long int c_max;
	//int c_media=0;
	short int aux1;
	short int aux2;
	//short int aux3;
	short int aux_p[10];
	short int op;
	short int t;
	int i,j, ite, flag, excess_temp;
	s[threadIdx.x].s = (short int*)malloc(sizeof(short int)*inst->nJobs);
	s[threadIdx.x].resUsage = (short int*)malloc(sizeof(short int)*inst->mAgents);
	hiprand_init(seed[threadIdx.x],threadIdx.x,0,&states[threadIdx.x]);
	s[threadIdx.x].costFinal = sol->costFinal;
	s[threadIdx.x].excess = sol->excess;
	if(threadIdx.x==1){
		printf("Custo da solucao inicial: %ld\n", s[threadIdx.x].costFinal + s[threadIdx.x].excess*10000);
	}
	for(i=0; i<inst->nJobs; i++)
	{
		s[threadIdx.x].s[i] = sol->s[i];
	}
	for(i=0; i<inst->mAgents; i++)
	{
		s[threadIdx.x].resUsage[i] = sol->resUsage[i];
	}
	L_c = hiprand(&states[threadIdx.x])%101 + 50;
	B_c = sol->costFinal;
	N_c=0;
	ite=0;
	while(ite<=100000)
	{
		//do
		//{
			op = hiprand(&states[threadIdx.x])%2;
			//printf("custo final temp: %d\n", s[threadIdx.x].costFinal);
			aux=0;
			excess_temp = 0;
			// op = 1;
			if(op == 1)
			{
				delta=0;
				aux1 = hiprand(&states[threadIdx.x])%inst->nJobs;
				aux2 = hiprand(&states[threadIdx.x])%inst->mAgents;
				delta = inst->cost[aux1*inst->mAgents+aux2] - inst->cost[aux1*inst->mAgents + s[threadIdx.x].s[aux1]];
				if(s[threadIdx.x].resUsage[aux2] + inst->resourcesAgent[aux1*inst->mAgents+aux2] > inst->capacity[aux2]){
					excess_temp += s[threadIdx.x].resUsage[aux2] + inst->resourcesAgent[aux1*inst->mAgents+aux2] - inst->capacity[aux2];
				}
				if(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] - inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1]] > inst->capacity[s[threadIdx.x].s[aux1]]){
					excess_temp += s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] - inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1]] - inst->capacity[s[threadIdx.x].s[aux1]];
				}
				//{
				//	aux=1;
				//}
			}
			else
			{
				delta=0;
				aux = 1;
				t = hiprand(&states[threadIdx.x])%8 + 2;
				aux_p[0] = hiprand(&states[threadIdx.x])%inst->nJobs;
				delta -= inst->cost[ aux_p[0]*inst->mAgents+s[threadIdx.x].s[aux_p[0]]];
				for(i=1; i<=t; i++)
				{

					aux_p[i] = hiprand(&states[threadIdx.x])%inst->nJobs;
					aux1 = aux_p[i];
					aux_p[t]= inst->nJobs-1;
					do{
						flag = 0;
						for(j=0; j<i; j++)
						{
							if(aux_p[i]==aux_p[j])
							{
								flag = 1;
							}
						}
						if((s[threadIdx.x].s[aux_p[i]] != s[threadIdx.x].s[aux_p[i-1]])&&( s[threadIdx.x].s[aux_p[0]] != s[threadIdx.x].s[aux_p[t]] ) &&(flag!=1)/*&&(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] - inst->resourcesAgent[aux_p[i]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]] + inst->resourcesAgent[aux_p[i-1]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]] <= inst->capacity[s[threadIdx.x].s[aux_p[i]]])*/){
							break;
						}
						aux_p[i]=(aux_p[i]+1)%(inst->nJobs);
					}while(aux_p[i]!=aux1);
					if(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] - inst->resourcesAgent[aux_p[i]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]] + inst->resourcesAgent[aux_p[i-1]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]] > inst->capacity[s[threadIdx.x].s[aux_p[i]]]){
						excess_temp += s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] - inst->resourcesAgent[aux_p[i]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]] + inst->resourcesAgent[aux_p[i-1]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]] - inst->capacity[s[threadIdx.x].s[aux_p[i]]];
					}
					if(aux1==aux_p[i]){
						aux=0;
					}


					/*while((i==t)&&(s[threadIdx.x].s[aux_p[0]]==s[threadIdx.x].s[aux_p[t]]))
						{
							aux_p[i] = hiprand(&states[threadIdx.x])%inst->nJobs;
						}
						for(j=0; j<i; j++)
						{
							if(aux_p[i]==aux_p[j])
							{
								flag = 1;
							}
						}*/
					delta += inst->cost[aux_p[i-1]*inst->mAgents+s[threadIdx.x].s[aux_p[i]]];
					delta -= inst->cost[aux_p[i]*inst->mAgents+s[threadIdx.x].s[aux_p[i]]];


				}
				delta += inst->cost[aux_p[t]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]];
				if(s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[0]]] - inst->resourcesAgent[aux_p[0]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]] + inst->resourcesAgent[aux_p[t]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]]>inst->capacity[s[threadIdx.x].s[aux_p[0]]])
				{
					excess_temp += s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[0]]] - inst->resourcesAgent[aux_p[0]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]] + inst->resourcesAgent[aux_p[t]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]]-inst->capacity[s[threadIdx.x].s[aux_p[0]]];
				//	aux=0;
				}
			}
			test_ite++;
		//}
		//while(aux==0);
		if(test_ite> max_ite){
			max_ite = test_ite;
		}
		test_ite = 0;
		//excess_temp = 0;
		for(i=0;i<inst->mAgents;i++){
				 if(s[threadIdx.x].resUsage[i]-inst->capacity[i]>0){
					 excess_temp += s[threadIdx.x].resUsage[i]-inst->capacity[i];
				 }
		}


		if ((s[threadIdx.x].costFinal + delta + excess_temp*10000 < B_c)||(s[threadIdx.x].costFinal + delta + excess_temp*10000<= s[threadIdx.x].costFinal + s[threadIdx.x].excess*10000))
		{
			s[threadIdx.x].costFinal += delta;
			if(op==1)
			{
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux1]] -= inst->resourcesAgent[aux1*inst->mAgents + s[threadIdx.x].s[aux1] ];
				s[threadIdx.x].resUsage[aux2] += inst->resourcesAgent[aux1*inst->mAgents + aux2];
				s[threadIdx.x].s[aux1] = aux2;
			}
			else
			{
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[0]]] += inst->resourcesAgent[aux_p[t]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]];
				s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[0]]] -= inst->resourcesAgent[aux_p[0]*inst->mAgents + s[threadIdx.x].s[aux_p[0]]];
				aux = s[threadIdx.x].s[aux_p[0]];
				for(i=1; i<=t; i++)
				{
					s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] += inst->resourcesAgent[aux_p[i-1]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]];
					s[threadIdx.x].resUsage[s[threadIdx.x].s[aux_p[i]]] -= inst->resourcesAgent[aux_p[i]*inst->mAgents + s[threadIdx.x].s[aux_p[i]]];
					s[threadIdx.x].s[aux_p[i-1]] = s[threadIdx.x].s[aux_p[i]];
				}
				s[threadIdx.x].s[aux_p[t]] = aux;
			}
			s[threadIdx.x].excess = 0;
			for(i=0;i<inst->mAgents;i++){
					 if(s[threadIdx.x].resUsage[i]-inst->capacity[i]>0){
						 s[threadIdx.x].excess += s[threadIdx.x].resUsage[i]-inst->capacity[i];
					 }
			}
		}
		N_c++;
		if(N_c >= L_c)
		{
			B_c = s[threadIdx.x].costFinal + s[threadIdx.x].excess*1000;
			N_c = 0;
		}
		ite++;
	}

	if(threadIdx.x < 1)
	{
		c_min = s[threadIdx.x].costFinal + s[threadIdx.x].excess*1000;
		c_max = s[threadIdx.x].costFinal;
		for(i=0; i<nThreads; i++)
		{
			for(j=0;j<inst->nJobs;j++){
				atomicInc(&rank[j * inst->mAgents +s[i].s[j]],nThreads+1);
			}

			if(s[i].costFinal + s[i].excess*10000< c_min)
			{
 				c_min = s[i].costFinal + s[i].excess*10000;
				sol->costFinal = s[i].costFinal;
				for(j=0; j<inst->nJobs; j++)
				{
					sol->s[j] = s[i].s[j] ;
				}
				for(j=0; j<inst->mAgents; j++)
				{
					sol->resUsage[j] = s[i].resUsage[j];
				}
				sol->excess = 0;
				for(i=0;i<inst->mAgents;i++){
						 if(sol->resUsage[i]-inst->capacity[i]>0){
							sol->excess += sol->resUsage[i]-inst->capacity[i];
						 }
				}


			}
			if(s[i].costFinal>c_max){
				c_max = s[i].costFinal + s[i].excess*10000;
			}
			//c_media+=s[i].costFinal;
		}
		printf("\n%ld ---- ", c_min);
		printf("%d ----", max_ite);
		//c_media=c_media/nThreads;
		printf("%ld ---- ", c_max);
		//printf("%d ---- ", c_media);

	}

	free(s[threadIdx.x].s);
	free(s[threadIdx.x].resUsage);

}


Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents)
{
	//printf("Begin createGpuSolution!\n");

	size_t size_solution = sizeof(Solution)
                        				   + sizeof(Ts)*nJobs //vector s
                        				   + sizeof(TresUsage)*mAgents; // vector resUsage
	Solution *d_sol;
	gpuMalloc((void**)&d_sol, size_solution);
	//printf("malloc solution ok!\n");
	//getchar();
	gpuMemset(d_sol,0,size_solution);
	//printf("memset Solution ok!\n");
	//getchar();

	h_solution->s = (Ts*)(d_sol+1);
	h_solution->resUsage = (TresUsage*)(h_solution->s + nJobs);

	//printf("adjusting solution GPU pointers\n");
	//getchar();

	gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);

	//printf("memcpy Solution ok!\n");
	//getchar();

	return d_sol;

}

